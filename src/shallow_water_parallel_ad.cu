#include "hip/hip_runtime.h"
// ---------------------------------------------------------
// TDT4200 Parallel Computing - Graded CUDA
// ---------------------------------------------------------
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <unistd.h>
#include <math.h>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

#include "../inc/argument_utils.h"


typedef int64_t int_t;
typedef double real_t;

int_t
    N,
    max_iteration,
    snapshot_frequency;

const real_t
    domain_size = 10.0,
    gravity = 9.81,
    density = 997.0;

real_t
    *h_mass_0 = NULL,
    *h_mass_1 = NULL,
    *d_mass_0 = NULL,
    *d_mass_1 = NULL,

    *h_mass_velocity_x_0 = NULL,
    *h_mass_velocity_x_1 = NULL,
    *d_mass_velocity_x_0 = NULL,
    *d_mass_velocity_x_1 = NULL,

    *h_mass_velocity_y_0 = NULL,
    *h_mass_velocity_y_1 = NULL,
    *d_mass_velocity_y_0 = NULL,
    *d_mass_velocity_y_1 = NULL,

    *h_mass_velocity = NULL,
    *d_mass_velocity = NULL,

    *h_velocity_x = NULL,
    *d_velocity_x = NULL,
    *h_velocity_y = NULL,
    *d_velocity_y = NULL,

    *h_acceleration_x = NULL,
    *d_acceleration_x = NULL,
    *h_acceleration_y = NULL,
    *d_acceleration_y = NULL,
    dx,
    dt;

#define PN(y,x)         mass_0[(y)*(N+2)+(x)]
#define PN_next(y,x)    mass_1[(y)*(N+2)+(x)]
#define PNU(y,x)        mass_velocity_x_0[(y)*(N+2)+(x)]
#define PNU_next(y,x)   mass_velocity_x_1[(y)*(N+2)+(x)]
#define PNV(y,x)        mass_velocity_y_0[(y)*(N+2)+(x)]
#define PNV_next(y,x)   mass_velocity_y_1[(y)*(N+2)+(x)]
#define PNUV(y,x)       mass_velocity[(y)*(N+2)+(x)]
#define U(y,x)          velocity_x[(y)*(N+2)+(x)]
#define V(y,x)          velocity_y[(y)*(N+2)+(x)]
#define DU(y,x)         acceleration_x[(y)*(N+2)+(x)]
#define DV(y,x)         acceleration_y[(y)*(N+2)+(x)]


#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) {
        fprintf(
            stderr,
            "GPUassert: \"%s | code: %d\"\n%s %d\n", 
            hipGetErrorString(code), (int) code, file, line);
        if (abort) exit(code);
    }
}

__global__ void time_step( 
    real_t *velocity_x,
    real_t *velocity_y,
    real_t *acceleration_x, 
    real_t *acceleration_y,
    real_t *mass_velocity_x_0, 
    real_t *mass_velocity_x_1,
    real_t *mass_velocity_y_0, 
    real_t *mass_velocity_y_1,
    real_t *mass_velocity, 
    real_t *mass_0, 
    real_t *mass_1,
    real_t dx, 
    real_t dt,
    int_t N
 );


// TODO: Rewrite boundary_condition as a device function.
__device__ void 
boundary_condition ( real_t *domain_variable, int sign, int_t N );

void domain_init ( void );
void domain_save ( int_t iteration );
void domain_finalize ( void );

void *domain_save_threaded ( void *iter );

void
swap ( real_t** t1, real_t** t2 )
{
    real_t* tmp;
	tmp = *t1;
	*t1 = *t2;
	*t2 = tmp;
}


int
main ( int argc, char **argv )
{

    OPTIONS *options = parse_args( argc, argv );
    if ( !options )
    {
        fprintf( stderr, "Argument parsing failed\n" );
        exit(1);
    }

    N = options->N;
    max_iteration = options->max_iteration;
    snapshot_frequency = options->snapshot_frequency;

    domain_init();

    uint grid_size = (unsigned int) ceil((double)(N+2) / (double) 32.0);
    dim3 grid_layout = {grid_size, grid_size, 1};
    dim3 block_layout = {32, 32, 1};

    int elements = (N+2)*(N+2);

    for ( int_t iteration = 0; iteration <= max_iteration; iteration++ )
    {
        // TODO: Launch time_step kernels

        void * kernel_args[] = { 
            (void*) &d_velocity_x, 
            (void*) &d_velocity_y, 
            (void*) &d_acceleration_x,
            (void*) &d_acceleration_y,
            (void*) &d_mass_velocity_x_0,
            (void*) &d_mass_velocity_x_1,
            (void*) &d_mass_velocity_y_0,
            (void*) &d_mass_velocity_y_1,
            (void*) &d_mass_velocity,
            (void*) &d_mass_0,
            (void*) &d_mass_1,
            (void*) &dx,
            (void*) &dt,
            (void*) &N, 
        };

        // launch kernel in cooperative with appropriate args
        cudaErrorCheck(
            hipLaunchCooperativeKernel(
                (void*) time_step,  
                grid_layout, 
                block_layout,
                kernel_args
            )
        );


        if ( iteration % snapshot_frequency == 0 )
        {
            printf (
                "Iteration %ld of %ld, (%.2lf%% complete)\n",
                iteration,
                max_iteration,
                100.0 * (real_t) iteration / (real_t) max_iteration
            );

            // TODO: Copy the masses from the device to host prior to domain_save
            cudaErrorCheck(hipMemcpy(h_mass_0, d_mass_0, elements*sizeof(real_t), hipMemcpyDeviceToHost));


            domain_save ( iteration );
        }

        // TODO: Swap device buffer pointers between iterations

        swap ( &d_mass_0, &d_mass_1 );
        swap ( &d_mass_velocity_x_0, &d_mass_velocity_x_1 );
        swap ( &d_mass_velocity_y_0, &d_mass_velocity_y_1 );
    }

    domain_finalize();

    exit ( EXIT_SUCCESS );
}

__global__ void
time_step( 
    real_t *velocity_x,
    real_t *velocity_y,
    real_t *acceleration_x, 
    real_t *acceleration_y,
    real_t *mass_velocity_x_0, 
    real_t *mass_velocity_x_1,
    real_t *mass_velocity_y_0, 
    real_t *mass_velocity_y_1,
    real_t *mass_velocity, 
    real_t *mass_0, 
    real_t *mass_1,
    real_t dx, 
    real_t dt,
    int_t N
)
{
    // TODO: Rewrite this function as one or more CUDA kernels
    // ---------------------------------------------------------
    // To ensure correct results, the participating threads in the thread
    // grid must be synchronized after calculating the accelerations (DU, DV).
    // If the grid is not synchronized, data dependencies cannot be guaranteed.


    cg::thread_group g = cg::this_thread();
    
    // time_step_1 

    boundary_condition ( mass_0, 1, N );
    boundary_condition ( mass_velocity_x_0, -1, N );
    boundary_condition ( mass_velocity_y_0, -1, N );

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;


    if ((0 <= y && y <= N+1) && (0 <= x && x <= N+1)) {

        DU(y,x) = PN(y,x) * U(y,x) * U(y,x)
                + 0.5 * gravity * ( PN(y,x) * PN(y,x) / density );
        DV(y,x) = PN(y,x) * V(y,x) * V(y,x)
                + 0.5 * gravity * ( PN(y,x) * PN(y,x) / density );
    
    }

    cg::sync(g);

    // time_step_2    

    if ((1 <= y && y <= N) && (1 <= x && x <= N)) {

        U(y,x) = PNU(y,x) / PN(y,x);
        V(y,x) = PNV(y,x) / PN(y,x);

        PNUV(y,x) = PN(y,x) * U(y,x) * V(y,x);

        PNU_next(y,x) = 0.5*( PNU(y,x+1) + PNU(y,x-1) ) - dt*(
                        ( DU(y,x+1) - DU(y,x-1) ) / (2*dx)
                        + ( PNUV(y,x+1) - PNUV(y,x-1) ) / (2*dx));

        PNV_next(y,x) = 0.5*( PNV(y+1,x) + PNV(y-1,x) ) - dt*(
                        ( DV(y+1,x) - DV(y-1,x) ) / (2*dx)
                        + ( PNUV(y+1,x) - PNUV(y-1,x) ) / (2*dx));

        PN_next(y,x) = 0.25*( PN(y,x+1) + PN(y,x-1) + PN(y+1,x) + PN(y-1,x) ) - dt*(
                    ( PNU(y,x+1) - PNU(y,x-1) ) / (2*dx)
                    + ( PNV(y+1,x) - PNV(y-1,x) ) / (2*dx));
    }
}


// TODO: Rewrite boundary_condition as a device function.
__device__ void
boundary_condition ( real_t *domain_variable, int sign, int_t N )
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    #define VAR(y,x) domain_variable[(y)*(N+2)+(x)]
    VAR(   0, 0   ) = sign*VAR(   2, 2   );
    VAR( N+1, 0   ) = sign*VAR( N-1, 2   );
    VAR(   0, N+1 ) = sign*VAR(   2, N-1 );
    VAR( N+1, N+1 ) = sign*VAR( N-1, N-1 );

    if (1 <= y && y <= N)  {
        VAR(   y, 0   ) = sign*VAR(   y, 2   );
        VAR(   y, N+1 ) = sign*VAR(   y, N-1 );
    }
    if (1 <= x && x <= N) {
        VAR(   0, x   ) = sign*VAR(   2, x   );
        VAR( N+1, x   ) = sign*VAR( N-1, x   );
    }
    #undef VAR
}


void
domain_init ( void )
{
    int elements = (N+2)*(N+2);

    // TODO: Allocate device buffers for masses, velocities and accelerations.
    // -----------------------------------------------------
    h_mass_0 = (real_t *) calloc ( elements, sizeof(real_t) );
    h_mass_1 = (real_t *) calloc ( elements, sizeof(real_t) );

    h_mass_velocity_x_0 = (real_t *) calloc ( elements, sizeof(real_t) );
    h_mass_velocity_x_1 = (real_t *) calloc ( elements, sizeof(real_t) );
    h_mass_velocity_y_0 = (real_t *) calloc ( elements, sizeof(real_t) );
    h_mass_velocity_y_1 = (real_t *) calloc ( elements, sizeof(real_t) );

    h_mass_velocity = (real_t *) calloc ( elements, sizeof(real_t) );

    h_velocity_x =  (real_t *) calloc ( elements, sizeof(real_t) );
    h_velocity_y = (real_t *) calloc ( elements, sizeof(real_t) );
    h_acceleration_x = (real_t *) calloc ( elements, sizeof(real_t) );
    h_acceleration_y = (real_t *) calloc ( elements, sizeof(real_t) );

    cudaErrorCheck(hipMalloc(&d_mass_0, elements * sizeof(real_t)));
    hipMalloc(&d_mass_1, elements * sizeof(real_t));

    hipMalloc(&d_mass_velocity_x_0, elements * sizeof(real_t));
    hipMalloc(&d_mass_velocity_x_1, elements * sizeof(real_t));
    hipMalloc(&d_mass_velocity_y_0, elements * sizeof(real_t));
    hipMalloc(&d_mass_velocity_y_1, elements * sizeof(real_t));

    hipMalloc(&d_mass_velocity, elements * sizeof(real_t));

    hipMalloc(&d_velocity_x, elements * sizeof(real_t));
    hipMalloc(&d_velocity_y, elements * sizeof(real_t));

    hipMalloc(&d_acceleration_x, elements * sizeof(real_t));
    hipMalloc(&d_acceleration_y, elements * sizeof(real_t));



    for ( int_t y=1; y<=N; y++ )
    {
        for ( int_t x=1; x<=N; x++ )
        {
	    h_mass_0[y*(N+2) + x] = 1e-3;
	    h_mass_velocity_x_0[y*(N+2) + x] = 0.0;
	    h_mass_velocity_y_0[y*(N+2) + x] = 0.0;

            real_t cx = x-N/2;
            real_t cy = y-N/2;
            if ( sqrt ( cx*cx + cy*cy ) < N/20.0 )
            {
                h_mass_0[y*(N+2) + x] -= 5e-4*exp (
                    - 4*pow( cx, 2.0 ) / (real_t)(N)
                    - 4*pow( cy, 2.0 ) / (real_t)(N)
                );
            }

            h_mass_0[y*(N+2) + x] *= density;
        }
    }

    dx = domain_size / (real_t) N;
    dt = 5e-2;

    hipMemcpy(d_mass_0           , h_mass_0           , elements * sizeof(real_t), hipMemcpyHostToDevice );
    hipMemcpy(d_mass_1           , h_mass_1           , elements * sizeof(real_t), hipMemcpyHostToDevice );

    hipMemcpy(d_mass_velocity_x_0, h_mass_velocity_x_0, elements * sizeof(real_t), hipMemcpyHostToDevice );
    hipMemcpy(d_mass_velocity_x_1, h_mass_velocity_x_1, elements * sizeof(real_t), hipMemcpyHostToDevice );
    hipMemcpy(d_mass_velocity_y_0, h_mass_velocity_y_0, elements * sizeof(real_t), hipMemcpyHostToDevice );
    hipMemcpy(d_mass_velocity_y_1, h_mass_velocity_y_1, elements * sizeof(real_t), hipMemcpyHostToDevice );

    hipMemcpy(d_mass_velocity    , h_mass_velocity    , elements * sizeof(real_t), hipMemcpyHostToDevice );

    hipMemcpy(d_velocity_x       , h_velocity_x       , elements * sizeof(real_t), hipMemcpyHostToDevice );
    hipMemcpy(d_velocity_y       , h_velocity_y       , elements * sizeof(real_t), hipMemcpyHostToDevice );

    hipMemcpy(d_acceleration_x   , h_acceleration_x   , elements * sizeof(real_t), hipMemcpyHostToDevice );
    hipMemcpy(d_acceleration_y   , h_acceleration_y   , elements * sizeof(real_t), hipMemcpyHostToDevice );
}


void
domain_save ( int_t iteration )
{
    int_t index = iteration / snapshot_frequency;
    char filename[256];
    memset ( filename, 0, 256*sizeof(char) );
    sprintf ( filename, "data/%.5ld.bin", index );

    FILE *out = fopen ( filename, "wb" );
    if ( !out )
    {
        fprintf( stderr, "Failed to open file %s\n", filename );
        exit(1);
    }
    
    for ( int_t y = 1; y <= N; y++ )
    {
        fwrite ( &h_mass_0[y*(N+2)+1], N, sizeof(real_t), out );
    }
    fclose ( out );
}

void
domain_finalize ( void )
{
    free ( h_mass_0 );
    free ( h_mass_1 );
    free ( h_mass_velocity_x_0 );
    free ( h_mass_velocity_x_1 );
    free ( h_mass_velocity_y_0 );
    free ( h_mass_velocity_y_1 );
    free ( h_mass_velocity );
    free ( h_velocity_x );
    free ( h_velocity_y );
    free ( h_acceleration_x );
    free ( h_acceleration_y );

    // TODO: Free device arrays
    hipFree(d_mass_0);
    hipFree(d_mass_1);

    hipFree(d_mass_velocity_x_0);
    hipFree(d_mass_velocity_x_1);
    hipFree(d_mass_velocity_y_0);
    hipFree(d_mass_velocity_y_1);

    hipFree(d_mass_velocity);

    hipFree(d_velocity_x);
    hipFree(d_velocity_y);
    
    hipFree(d_acceleration_x);
    hipFree(d_acceleration_y);

}
